#include "hip/hip_runtime.h"
// Wave 2D GPU Cuda aware MPI
// nvcc -arch=sm_52 --compiler-bindir mpic++ --compiler-options -O3 MPI_Wave_2D_v3.cu
#include "stdio.h"
#include "stdlib.h"
#include "math.h"
#include "hip/hip_runtime.h"
#define NDIMS  2

#define USE_SINGLE_PRECISION    /* Comment this line using "//" if you want to use double precision.  */
#ifdef USE_SINGLE_PRECISION
#define DAT      float
#define PRECIS   4
#else
#define DAT      double
#define PRECIS   8
#endif
#define GPU_ID   3
#define OVERLENGTH_X  1
#define OVERLENGTH_Y  1
#define zeros(A,nx,ny)  DAT *A##_d,*A##_h; A##_h = (DAT*)malloc(((nx)*(ny))*sizeof(DAT)); \
                        for(i=0; i < ((nx)*(ny)); i++){ A##_h[i]=(DAT)0.0; }              \
                        hipMalloc(&A##_d      ,((nx)*(ny))*sizeof(DAT));                 \
                        hipMemcpy( A##_d,A##_h,((nx)*(ny))*sizeof(DAT),hipMemcpyHostToDevice);
#define free_all(A)     free(A##_h); hipFree(A##_d);
#define gather(A,nx,ny) hipMemcpy( A##_h,A##_d,((nx)*(ny))*sizeof(DAT),hipMemcpyDeviceToHost);
// --------------------------------------------------------------------- //
// Physics
const DAT Lx   = 40.0;
const DAT Ly   = 10.0;
const DAT k    = 1.0;
const DAT rho  = 1.0;
// Numerics
#define BLOCK_X  32
#define BLOCK_Y  32
#define GRID_X   24
#define GRID_Y   24
#define DIMS_X   8
#define DIMS_Y   2
const int nx = BLOCK_X*GRID_X - OVERLENGTH_X;
const int ny = BLOCK_Y*GRID_Y - OVERLENGTH_Y;
const int nt = 2200;
// Preprocessing
DAT    dx, dy;
size_t Nix, Niy, Niz;
// GPU
#include "geocomp_unil_mpi2D.h"

// Computing physics kernels /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void init(DAT* x, DAT* y, DAT* P, int* coords, const DAT Lx, const DAT Ly, DAT dx, DAT dy, const int nx, const int ny){
    int ix = blockIdx.x*blockDim.x + threadIdx.x; // thread ID, dimension x
    int iy = blockIdx.y*blockDim.y + threadIdx.y; // thread ID, dimension y

    if (iy<ny && ix<nx){ x[ix + iy*nx] = (DAT)(coords[0]*(nx-2) + ix)*dx - (DAT)0.5*Lx; }
    if (iy<ny && ix<nx){ y[ix + iy*nx] = (DAT)(coords[1]*(ny-2) + iy)*dy - (DAT)0.5*Ly; }
    if (iy<ny && ix<nx){ P[ix + iy*nx] = exp(-(x[ix + iy*nx]*x[ix + iy*nx]) -(y[ix + iy*nx]*y[ix + iy*nx])); }
}
__global__ void compute_V(DAT* Vx, DAT* Vy, DAT* P, DAT dt, const DAT rho, DAT dx, DAT dy, const int nx, const int ny){
    int ix = blockIdx.x*blockDim.x + threadIdx.x; // thread ID, dimension x
    int iy = blockIdx.y*blockDim.y + threadIdx.y; // thread ID, dimension y

    if (iy<ny && ix>0 && ix<nx){
        Vx[ix + iy*(nx+1)] = Vx[ix + iy*(nx+1)] - (dt/dx/rho)*(P[ix + iy*nx]-P[ix-1 +  iy   *nx]); }
    if (iy>0 && iy<ny && ix<nx){
        Vy[ix + iy*(nx  )] = Vy[ix + iy*(nx  )] - (dt/dy/rho)*(P[ix + iy*nx]-P[ix   + (iy-1)*nx]); }
}
__global__ void compute_P(DAT* Vx, DAT* Vy, DAT* P, DAT dt, const DAT k, DAT dx, DAT dy, const int nx, const int ny){
    int ix = blockIdx.x*blockDim.x + threadIdx.x; // thread ID, dimension x
    int iy = blockIdx.y*blockDim.y + threadIdx.y; // thread ID, dimension y
    
    if (iy<ny && ix<nx){
        P[ix + iy*nx] = P[ix + iy*nx] - dt*k*(((DAT)1.0/dx)*(Vx[(ix+1) + iy    *(nx+1)]-Vx[ix + iy*(nx+1)]) 
                                            + ((DAT)1.0/dy)*(Vy[ ix    + (iy+1)* nx   ]-Vy[ix + iy* nx   ]) ); }
}

int main(int argc, char *argv[]){
    int i, it;
    set_up_grid();
    set_up_parallelisation();
    // Initial arrays
    zeros(x  ,nx  ,ny  );
    zeros(y  ,nx  ,ny  );
    zeros(P  ,nx  ,ny  );
    zeros(Vx ,nx+1,ny  );
    zeros(Vy ,nx  ,ny+1);
    // MPI sides    
    init_sides(Vx ,nx+1,ny  );
    init_sides(Vy ,nx  ,ny+1);
    // Preprocessing
    Nix  = ((nx-2)*dims[0])+2;
    Niy  = ((ny-2)*dims[1])+2;
    dx   = Lx/((DAT)Nix-(DAT)1.0);  // Global dx, dy
    dy   = Ly/((DAT)Niy-(DAT)1.0);
    DAT dt = min(dx,dy)/sqrt(k/rho)/4.1;
    // Initial conditions
    init<<<grid,block>>>(x_d, y_d, P_d, coords_d, Lx, Ly, dx, dy, nx, ny);    hipDeviceSynchronize();
    // Action
    for (it=0;it<nt;it++){
        if (it==3){ tic(); }
        if (it>0){
        compute_P<<<grid,block>>>(Vx_d, Vy_d, P_d, dt, k,   dx, dy, nx, ny);  hipDeviceSynchronize(); }
        compute_V<<<grid,block>>>(Vx_d, Vy_d, P_d, dt, rho, dx, dy, nx, ny);  hipDeviceSynchronize();
        update_sides(Vx,nx+1,ny  );
        update_sides(Vy,nx  ,ny+1);
    }//it
    tim("Performance", Nix*Niy*(nt-3)*6*PRECIS/(1e9)); // timer test
    // printf("Process %d used GPU with id %d.\n",me,gpu_id);
    free_all(x );
    free_all(y );
    free_all(P );
    free_all(Vx);
    free_all(Vy);
    // MPI
    free_sides(Vx);
    free_sides(Vy);
    
    clean_cuda();
    MPI_Finalize();
    return 0;
}
